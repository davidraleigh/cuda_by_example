#include "hip/hip_runtime.h"
#include "../common/book.h"

#define N 10

__global__ void add(int *a, int *b, int *c) {
	int tid = threadIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(void) {
	int h_a[N], h_b[N], h_c[N];
	int *d_a, *d_b, *d_c;

	for (int i = 0; i < N; i++)
	{
		h_a[i] = i;
		h_b[i] = i * i;
	}

	HANDLE_ERROR(hipMalloc((void**)&d_a, N * sizeof(int)));	
	HANDLE_ERROR(hipMalloc((void**)&d_b, N* sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_c, N* sizeof(int)));

	HANDLE_ERROR(hipMemcpy(d_a, h_a, N*sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_b, h_b, N*sizeof(int), hipMemcpyHostToDevice));
	add << <1, N >> >(d_a, d_b, d_c);

	HANDLE_ERROR(hipMemcpy(h_c, d_c, N*sizeof(int), hipMemcpyDeviceToHost));

	for (int i = 0; i < N; i++)
	{
		printf("results of a + b = c:\n%d + %d = %d\n\n", h_a[i], h_b[i], h_c[i]);
	}
	return 0;

	HANDLE_ERROR(hipFree(d_a));
	HANDLE_ERROR(hipFree(d_b));
	HANDLE_ERROR(hipFree(d_c));
};